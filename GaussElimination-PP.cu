#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include "hip/hip_runtime.h"
#include <string.h>

#define MAXBLOCKSIZE 512

int Size;
float *a, *b, *finalVec;
float *m;

FILE *fp;
void ForwardSub();
void BackSub();
void checkCUDAError(const char *msg);

void InitPerRun() 
{
	int i;
	for (i=0; i<Size*Size; i++)
			*(m+i) = 0.0;
}


void InitMat(float *ary, int nrow, int ncol)
{
	int i, j;
	
	for (i=0; i<nrow; i++) {
		for (j=0; j<ncol; j++) {
			fscanf(fp, "%f",  ary+Size*i+j);
		}
	}  
}

void InitAry(float *ary, int ary_size)
{
	int i;
	
	for (i=0; i<ary_size; i++) {
		fscanf(fp, "%f",  &ary[i]);
	}
}

void PrintMat(float *ary, int nrow, int ncol)
{
	int i, j;
	
	for (i=0; i<nrow; i++) {
		for (j=0; j<ncol; j++) {
			printf("%8.2f ", *(ary+Size*i+j));
		}
		printf("\n");
	}
	printf("\n");
}

__global__ void Fan1(float *m_cuda, float *a_cuda, int Size, int t)
{   
	

	if(threadIdx.x + blockIdx.x * blockDim.x >= Size-1-t) return;
	*(m_cuda+Size*(blockDim.x*blockIdx.x+threadIdx.x+t+1)+t) = *(a_cuda+Size*(blockDim.x*blockIdx.x+threadIdx.x+t+1)+t) / *(a_cuda+Size*t+t);
}

__global__ void Fan2(float *m_cuda, float *a_cuda, float *b_cuda,int Size, int j1, int t)
{
	if(threadIdx.x + blockIdx.x * blockDim.x >= Size-1-t) return;
	if(threadIdx.y + blockIdx.y * blockDim.y >= Size-t) return;
	
	int xidx = blockIdx.x * blockDim.x + threadIdx.x;
	int yidx = blockIdx.y * blockDim.y + threadIdx.y;
	
	
	a_cuda[Size*(xidx+1+t)+(yidx+t)] -= m_cuda[Size*(xidx+1+t)+t] * a_cuda[Size*t+(yidx+t)];
	
	if(yidx == 0){
		
		b_cuda[xidx+1+t] -= m_cuda[Size*(xidx+1+t)+(yidx+t)] * b_cuda[t];
	}
}




void ForwardSub()
{
	int t;
    float *m_cuda,*a_cuda,*b_cuda;
	
	
	hipMalloc((void **) &m_cuda, Size * Size * sizeof(float));
	 
	hipMalloc((void **) &a_cuda, Size * Size * sizeof(float));
	
	hipMalloc((void **) &b_cuda, Size * sizeof(float));	

	
	hipMemcpy(m_cuda, m, Size * Size * sizeof(float),hipMemcpyHostToDevice );
	hipMemcpy(a_cuda, a, Size * Size * sizeof(float),hipMemcpyHostToDevice );
	hipMemcpy(b_cuda, b, Size * sizeof(float),hipMemcpyHostToDevice );
	
	int block_size,grid_size;
	
	block_size = MAXBLOCKSIZE;
	grid_size = (Size/block_size) + (!(Size%block_size)? 0:1);
	


	dim3 dimBlock(block_size);
	dim3 dimGrid(grid_size);
	
	
	int blockSize2d, gridSize2d;
	blockSize2d = 4;
	gridSize2d = (Size/blockSize2d) + (!(Size%blockSize2d?0:1)); 
	
	dim3 dimBlockXY(blockSize2d,blockSize2d);
	dim3 dimGridXY(gridSize2d,gridSize2d);

   
    struct timeval time_start;
    gettimeofday(&time_start, NULL);
	for (t=0; t<(Size-1); t++) {
		Fan1<<<dimGrid,dimBlock>>>(m_cuda,a_cuda,Size,t);
		hipDeviceSynchronize();
		Fan2<<<dimGridXY,dimBlockXY>>>(m_cuda,a_cuda,b_cuda,Size,Size-t,t);
		hipDeviceSynchronize();
		checkCUDAError("Fan2");
	}
	
	struct timeval time_end;
    gettimeofday(&time_end, NULL);
   // totalKernelTime = (time_end.tv_sec * 1000000 + time_end.tv_usec) - (time_start.tv_sec * 1000000 + time_start.tv_usec);
	
	
	hipMemcpy(m, m_cuda, Size * Size * sizeof(float),hipMemcpyDeviceToHost );
	hipMemcpy(a, a_cuda, Size * Size * sizeof(float),hipMemcpyDeviceToHost );
	hipMemcpy(b, b_cuda, Size * sizeof(float),hipMemcpyDeviceToHost );
	hipFree(m_cuda);
	hipFree(a_cuda);
	hipFree(b_cuda);
}

void BackSub()
{
	
	finalVec = (float *) malloc(Size * sizeof(float));
	
	int i,j;
	for(i=0;i<Size;i++){
		finalVec[Size-i-1]=b[Size-i-1];
		for(j=0;j<i;j++)
		{
			finalVec[Size-i-1]-=*(a+Size*(Size-i-1)+(Size-j-1)) * finalVec[Size-j-1];
		}
		finalVec[Size-i-1]=finalVec[Size-i-1]/ *(a+Size*(Size-i-1)+(Size-i-1));
	}
}



void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, 
                                  hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }                         
}


void PrintAry(float *ary, int ary_size)
{
	int i;
	for (i=0; i<ary_size; i++) {
		printf("%8.2f ", ary[i]);
	}
	printf("\n\n");
}
int main()
{
	fp=fopen("samplege.txt","r");
        fscanf(fp, "%d", &Size);	
	 
	a = (float *) malloc(Size * Size * sizeof(float));
	 
	InitMat(a, Size, Size);
	
	b = (float *) malloc(Size * sizeof(float));
	
	InitAry(b, Size);
	
		
	 m = (float *) malloc(Size * Size * sizeof(float));

	InitPerRun();

        printf("Matrix a is: \n");
        PrintMat(a, Size, Size);
	
	printf("Array b is: \n");
        PrintAry(b, Size);

	
	ForwardSub();
	BackSub();
	printf("The final solution is: \n");
        PrintAry(finalVec,Size);

	
        free(m);
        free(a);
        free(b);
          
}
